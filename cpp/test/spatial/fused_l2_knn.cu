/*
 * Copyright (c) 2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "../test_utils.h"

#include <faiss/gpu/GpuDistance.h>
#include <faiss/gpu/GpuResources.h>
#include <faiss/gpu/StandardGpuResources.h>
#include <faiss/utils/Heap.h>
#include <faiss/gpu/utils/Limits.cuh>
#include <faiss/gpu/utils/Select.cuh>

#include <raft/linalg/distance_type.h>
#include <raft/spatial/knn/detail/common_faiss.h>
#include <raft/random/rng.cuh>
#include <raft/spatial/knn/detail/fused_l2_knn.cuh>
#include <raft/spatial/knn/knn.hpp>

#include <rmm/device_buffer.hpp>

#include <gtest/gtest.h>

#include <cstddef>
#include <iostream>
#include <vector>

namespace raft {
namespace spatial {
namespace knn {
struct FusedL2KNNInputs {
  int num_queries;
  int num_db_vecs;
  int dim;
  int k;
  raft::distance::DistanceType metric_;
};

template <typename T>
class FusedL2KNNTest : public ::testing::TestWithParam<FusedL2KNNInputs> {
 protected:
  void testBruteForce() {
    hipStream_t stream = handle_.get_stream();

    detail::fusedL2Knn(dim, raft_indices_, raft_distances_, database,
                       search_queries, num_db_vecs, num_queries, k_, true, true,
                       stream, metric);

    launchFaissBfknn();
    // Only verifying indices.
    ASSERT_TRUE(devArrMatchInRange(faiss_indices_, raft_indices_, num_queries,
                                   k_, raft::Compare<int64_t>(), stream));
  }

  void SetUp() override {
    params_ = ::testing::TestWithParam<FusedL2KNNInputs>::GetParam();
    num_queries = params_.num_queries;
    num_db_vecs = params_.num_db_vecs;
    dim = params_.dim;
    k_ = params_.k;
    metric = params_.metric_;

    hipStream_t stream = handle_.get_stream();

    raft::allocate(database, num_db_vecs * dim, stream, true);
    raft::allocate(search_queries, num_queries * dim, stream, true);

    unsigned long long int seed = 1234ULL;
    raft::random::Rng r(seed);
    r.uniform(database, num_db_vecs * dim, T(-1.0), T(1.0), stream);
    r.uniform(search_queries, num_queries * dim, T(-1.0), T(1.0), stream);

    raft::allocate(raft_indices_, num_queries * k_, stream, true);
    raft::allocate(raft_distances_, num_queries * k_, stream, true);
    raft::allocate(faiss_indices_, num_queries * k_, stream, true);
    raft::allocate(faiss_distances_, num_queries * k_, stream, true);
  }

  void TearDown() override {
    hipStream_t stream = handle_.get_stream();
    raft::deallocate_all(stream);
  }

  void launchFaissBfknn() {
    faiss::MetricType m = detail::build_faiss_metric(metric);

    faiss::gpu::StandardGpuResources gpu_res;

    gpu_res.noTempMemory();
    int device;
    CUDA_CHECK(hipGetDevice(&device));
    gpu_res.setDefaultStream(device, handle_.get_stream());

    faiss::gpu::GpuDistanceParams args;
    args.metric = m;
    args.metricArg = 0;
    args.k = k_;
    args.dims = dim;
    args.vectors = database;
    args.vectorsRowMajor = true;
    args.numVectors = num_db_vecs;
    args.queries = search_queries;
    args.queriesRowMajor = true;
    args.numQueries = num_queries;
    args.outDistances = faiss_distances_;
    args.outIndices = faiss_indices_;

    bfKnn(&gpu_res, args);
  }

 private:
  raft::handle_t handle_;
  FusedL2KNNInputs params_;
  int num_queries;
  int num_db_vecs;
  int dim;
  T *database;
  T *search_queries;
  int64_t *raft_indices_;
  T *raft_distances_;
  int64_t *faiss_indices_;
  T *faiss_distances_;
  int k_;
  raft::distance::DistanceType metric;
};

const std::vector<FusedL2KNNInputs> inputs = {
  {100, 1000, 16, 10, raft::distance::DistanceType::L2Expanded},
  {1000, 10000, 16, 10, raft::distance::DistanceType::L2Expanded},
  {100, 1000, 16, 50, raft::distance::DistanceType::L2Expanded},
  {20, 10000, 16, 10, raft::distance::DistanceType::L2Expanded},
  {1000, 10000, 16, 50, raft::distance::DistanceType::L2Expanded},
  {1000, 10000, 32, 50, raft::distance::DistanceType::L2Expanded},
  {10000, 40000, 32, 30, raft::distance::DistanceType::L2Expanded},
  // L2 unexpanded
  {100, 1000, 16, 10, raft::distance::DistanceType::L2Unexpanded},
  {1000, 10000, 16, 10, raft::distance::DistanceType::L2Unexpanded},
  {100, 1000, 16, 50, raft::distance::DistanceType::L2Unexpanded},
  {20, 10000, 16, 50, raft::distance::DistanceType::L2Unexpanded},
  {1000, 10000, 16, 50, raft::distance::DistanceType::L2Unexpanded},
  {1000, 10000, 32, 50, raft::distance::DistanceType::L2Unexpanded},
  {10000, 40000, 32, 30, raft::distance::DistanceType::L2Unexpanded}};

typedef FusedL2KNNTest<float> FusedL2KNNTestF;
TEST_P(FusedL2KNNTestF, FusedBruteForce) { this->testBruteForce(); }

INSTANTIATE_TEST_CASE_P(FusedL2KNNTest, FusedL2KNNTestF,
                        ::testing::ValuesIn(inputs));

}  // namespace knn
}  // namespace spatial
}  // namespace raft
